#include "hip/hip_runtime.h"
#include "SliceSM.h"

#include <iostream>
#include <assert.h>

#include "GM.h"
#include "SM.h"

using std::cout;
using std::endl;
using std::to_string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void sliceSM(int nbSlice,float* ptrPiHatGM);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

SliceSM::SliceSM(const Grid& grid , int nbSlice , double* ptrPiHat , bool isVerbose) :
	RunnableGPU(grid, "SliceSM-" + to_string(nbSlice), isVerbose), // classe parente
	//
	ptrPiHat(ptrPiHat), //
	nbSlice(nbSlice) //
    {
    this->ptrPiHatGM = NULL;
    this->sizeSM = grid.threadByBlock() * sizeof(float);

    // MM
	{
	GM::malloc0(&ptrPiHatGM, sizeof(float));
	}
    }

SliceSM::~SliceSM(void)
    {
    //MM
	{
	GM::free(ptrPiHatGM);
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void SliceSM::run()
    {
    // Etape 1 : lancer le kernel
    // Etape 2 : recuperer le resultat coter host (par exemple avec memcpyDToH_float)
    // Etape 3 : finaliser le calcul de PI

    // TODO SliceSM
    //1
    sliceSM<<<dg,db,sizeSM>>>(nbSlice , ptrPiHatGM);
    //2
    float result = 0.0;
    GM::memcpyDToH_float(&result, ptrPiHatGM);
    //3
    *ptrPiHat = (double)result;
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
