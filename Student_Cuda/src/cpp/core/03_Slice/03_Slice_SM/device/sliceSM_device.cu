#include "hip/hip_runtime.h"
#include "Thread2D.h"
#include "Thread1D.h"
#include "ReductionAdd.h"

#include "cudas.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void reductionIntraThread(float* tabSM,int nbSlice);
static __device__ float f(float x);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

__global__ void sliceSM(int nbSlice , float* ptrPiHatGM)
    {
    // Declarer tabSM
    extern __shared__ float tabSM[];

    // ReductionIntraThread
    reductionIntraThread(tabSM, nbSlice);

    __syncthreads();

    // Reduction de tabSM (use tools ReductionAdd)
    ReductionAdd::reduce(tabSM, ptrPiHatGM); // T* tabSM, T* ptrResultGM

    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * remplit la sm
 */
void reductionIntraThread(float* tabSM , int nbSlice)
    {
    const int TID = Thread1D::tid();
    const int TID_LOCAL = Thread1D::tidLocal();
    const int NB_THREAD = Thread1D::nbThread(); // DIFF entre nbthreadblock et nbthreadlocal ?
    int s = TID;

    const float DX = 1 / (float)nbSlice;
    float threadSum = 0;
    float xi;

    while (s < nbSlice)
	{
	xi = s * DX; // x value of function
	threadSum += f(xi);
	s += NB_THREAD;
	}
    tabSM[TID_LOCAL] = threadSum * DX;
    }

__device__ float f(float x)
    {
    return 4 / (1 + (x * x));
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

