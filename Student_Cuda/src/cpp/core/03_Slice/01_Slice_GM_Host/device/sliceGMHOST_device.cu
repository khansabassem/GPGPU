#include "hip/hip_runtime.h"
#include "Thread2D.h"
#include "Thread1D.h"
#include "cudas.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ float f(float x);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/**
 * <pre>
 * Chaque thread effecteur une reduction intrathread avec le patern d'entrelacement,
 * puis stocke son resultat dans SA case dans tabGM
 *
 * tabGM est un tableau promu, qui a autant de case que de thread
 * </pre>
 */
__global__ void reductionIntraThreadGMHOST(float* tabGM , int nbSlice) // grid 1d
    {
    const int NB_THREAD = Thread2D::nbThread();
    const int TID = Thread2D::tid();
    float threadSum = 0;
    int s = TID;
    const float DX = 1 / (float)nbSlice;
    float xi;
    while (s < nbSlice)
	{
	xi = s * DX; // x value of function
	threadSum += f(xi);
	s += NB_THREAD;
	}
    tabGM[TID] = threadSum;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ float f(float x)
    {
    return 4 / (1 + (x * x));
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

