#include "hip/hip_runtime.h"
#include "SliceGMHOST.h"

#include <iostream>
#include <assert.h>

#include "GM.h"
#include "Maths.h"
#include "Hardware.h"
#include "Kernel.h"

using std::cout;
using std::endl;
using std::to_string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/
// import du kernel
extern __global__ void reductionIntraThreadGMHOST(float* tabGM,int nbSlice);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

SliceGMHOST::SliceGMHOST(Grid grid , int nbSlice , double* ptrPiHat , bool isVerbose) :
	RunnableGPU(grid, "SliceGM-HOST-" + to_string(nbSlice), isVerbose), // classe parente
//
	nbSlice(nbSlice), //
	ptrPiHat(ptrPiHat) //
    {
    // ntabGM
	{
	this->nTabGM = grid.threadCounts();

	// Warning : Advanced : Et si plus de threads que slices? complique! (pas utile de le coder)
	}

    // MM
	{
	this->sizeTabGM = nTabGM * sizeof(float); // [octet]
	// step 0
	GM::malloc(&tabGM, sizeTabGM);
	}

    }

SliceGMHOST::~SliceGMHOST(void)
    {
    //step 4
	{
	GM::free(tabGM);
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

/**
 * <pre>
 * Idea globale
 *
 *	Etape 0 : Promotion d'un tableau en GM (MemoryManagement MM)		(Dans le constructeur)
 * 	Etape 1 : Reduction intra-thread dans un tableau promu en GM
 * 	Etape 2 : Copy du tableau coter host
 * 	Etape 3 : Reduction parallel du tableau coter host avec OMP
 * 	Etape 4 : Destruction GM						(Dans le destructeur)
 *
 * </pre>
 */
void SliceGMHOST::run()
    {
    // step 1
    //reductionIntraThreadGMHOST<<<dg,db>>>(tabGM, nbSlice);
    reductionIntraThreadGMHOST<<<grid.dg,grid.db>>>(tabGM, nbSlice);
    // step 2 et 3
    reductionHost();
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * Reduction paralle sur cpu du tableau promu en GM, ramener coter host
 */
void SliceGMHOST::reductionHost()
    {
    // 1) Creer un tableau sur la pile de bonne dimension
    // 2) Transferer la tabGM dedans
    // 3) reduction parallel avec OMP
    // 4) finalisation du calcul de ptrPiHat

    // 1
    float tab[nTabGM]; // chaque thread a une case pour mettre son résultat
    float const dx = 1 / (float)nbSlice;
    // 2
    GM::memcpyDToH(tab, tabGM, sizeTabGM);
    // 3
    float sum = 0;
#pragma omp parallel for reduction(+:sum)
    for (int i = 0; i < this->nTabGM; i++)
	{
	sum += tab[i];
	}
    // 4
    *ptrPiHat = sum * dx;
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
