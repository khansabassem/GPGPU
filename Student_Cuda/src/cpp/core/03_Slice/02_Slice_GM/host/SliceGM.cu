#include "hip/hip_runtime.h"
#include "SliceGM.h"

#include <iostream>
#include <assert.h>

#include "GM.h"
#include "Maths.h"
#include "Hardware.h"
#include "Kernel.h"

using std::cout;
using std::endl;
using std::to_string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void reductionIntraThreadGM(float* tabGM,int nbSlice);
extern __global__ void ecrasementGM(float* tabGM, int moitier);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

SliceGM::SliceGM(Grid grid , int nbSlice , double* ptrPiHat , bool isVerbose) :
	RunnableGPU(grid, "SliceGM-" + to_string(nbSlice), isVerbose), // classe parente
//
	nbSlice(nbSlice), //
	ptrPiHat(ptrPiHat) //
    {
    // ntabGM
	{
	this->nTabGM = grid.threadCounts();

	}

    // MM
	{
	this->sizeTabGM = nTabGM * sizeof(float); // [octet]
	// step 0
	GM::malloc(&tabGM, sizeTabGM);

	}
    }

SliceGM::~SliceGM(void)
    {
    //MM (device free)
	{
	GM::free(tabGM);
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

/**
 * <pre>
 * Idea globale
 *
 *	Etape 0 : Promotion d'un tableau en GM (MemoryManagement MM)
 * 	Etape 1 : Reduction intra-thread dans un tableau promu en GM
 * 	Etape 2 : Reduction du tableau en GM par ecrasement hierarchique 2 à 2
 * 		  On lance les kernels d'ecrasement depuis le host (chef d'orchestre)
 * 	Etape 4 : Copy du reseultat coter host
 * 	Etape 5 : Destruction GM
 * </pre>
 */
void SliceGM::run()
    {
    // step 1
    reductionIntraThreadGM<<<grid.dg,grid.db>>>(tabGM, nbSlice);
    // step 2 et 4
    reductionGM();

    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * Etape 1 : Lancement des kernels d'ecrasement depuis le host, dans une boucle,
 * Etape 2 : recuperer le resultat coter host
 * Etape 3 : finaliser le calcule de PI
 */
void SliceGM::reductionGM()
    {
    int middle = nTabGM >> 1; // nTabGM/2;
    float const dx = 1 / (float)nbSlice;
    //printf("middle: %d\n",middle);
    // Si la premiere grid n'est pas une puissance de 2... comment faire mtn ?
    dim3 dg = dim3(grid.dg.x, 1, 1);
    dim3 db = dim3(grid.db.x, 1, 1);
    while (middle > 0)
	{
	if (dg.x > 1)
	    dg.x = dg.x >> 1;
	else
	    db.x = db.x >> 1;
	ecrasementGM<<<dg,db>>>(tabGM, middle);
	middle = middle >> 1;

	}
    // 3 pas nécessaire de récuper tout le tableau, uniquement la dernière case.
    float result(0); //pourquoi est-il impossible de déclarer un pointeur ici ?

    GM::memcpyDToH_float(&result, tabGM); // on prend la première valeur donc ok

    // 4
    *ptrPiHat = result * dx;

    // Utiliser une autre grille que celle heriter de la classe parente dg, db
    // Votre grid ici doit avoir une taille speciale!
    // N'utiliser donc pas les vraibales dg et db de la super classe
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
