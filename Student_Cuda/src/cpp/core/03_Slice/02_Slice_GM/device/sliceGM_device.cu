#include "hip/hip_runtime.h"
#include "Thread2D.h"
#include "Thread1D.h"
#include "cudas.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ float f(float x);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/**
 * <pre>
 * Chaque thread effecteur une reduction intrathread avec le patern d'entrelacement,
 * puis stocke son résultat dans SA case dans tabGM
 *
 * tabGM est un tableau promu, qui a autant de case que de thread
 * </pre>
 */
__global__ void reductionIntraThreadGM(float* tabGM , int nbSlice)
    {
    const int NB_THREAD = Thread1D::nbThread();
    const int TID = Thread1D::tid();
    float threadSum = 0;
    int s = TID;
    const float DX = 1 / (float)nbSlice;
    float xi;
    while (s < nbSlice)
	{
	xi = s * DX; // x value of function
	threadSum += f(xi);
	s += NB_THREAD;
	}
    tabGM[TID] = threadSum;
    }

/**
 * <pre>
 * Effectue la reduction de tabGM cote device, par ecrasement 2 à 2 successif.
 * Ce kernel d ecrasement est appeler depuis le host dans une boucle, avec le bon nombre de thread
 *
 * Hypothese : |tabGM| est une puissance de 2
 *
 * Output: le resultat de la reduction est tans tabGM[0]
 * </pre>
 */
__global__ void ecrasementGM(float* tabGM , int middle)
    {
    const int TID = Thread1D::tid();
    tabGM[TID] += tabGM[TID + middle];
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ float f(float x)
    {
    return 4 / (1 + (x * x));
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

