#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

#include "Thread1D.h"
#include "cudas.h"
#include "entier.h"
#include "Lock.h"
#include "ReductionAdd.h"
#include "Reduction.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ static void reductionIntraThread(hiprandState* tabGeneratorGM , entier* tabSM , entier nbDarByThread , float h);
__device__ static float f(float x);

__device__ static long add(long x , long y);
__device__ static void addAtomic(long* ptrX , long y);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*-------------------*\
|* generateur alea   *|
 \*------------------*/

/**
 * Each thread gets same seed, a different sequence number, no offset
 * host side : Device::getDeviceId();
 */
__global__ void createGenerator(hiprandState* tabGeneratorGM , int deviceId)
    {
    // Customisation du generator:
    // Proposition, au lecteur de faire mieux !
    // Contrainte : Doit etre différent d'un GPU à l'autre
    // Contrainte : Doit etre différent d'un thread à l'autre

    const int TID = Thread1D::tid();
    int deltaSeed = deviceId * INT_MAX / 10000;
    int deltaSequence = deviceId * 100;
    int deltaOffset = deviceId * 100;
    int seed = 1234 + deltaSeed;
    int sequenceNumber = TID + deltaSequence;
    int offset = deltaOffset;

    hiprand_init(seed, sequenceNumber, offset, &tabGeneratorGM[TID]);
    }

/*-------------------*\
|* reduce 	     *|
 \*------------------*/

/**
 *
 * Chaque thread doit
 * 		1) tirer nbDarByThread et compter le nombre de dar sous la courbe
 * 		2) Le ranger sans sa case en tabSM
 * Puis on reduit les tabSM
 */
__global__ void montecarlo(hiprandState* tabGeneratorGM , entier nbDarByThread , entier* ptrNbDarUnderGM , float h)
    {
    // TODO Montecarlo
    extern __shared__ entier tabSM[];
    reductionIntraThread(tabGeneratorGM, tabSM, nbDarByThread, h);
    __syncthreads();

    // reduction des tabSM
	{
#ifdef DAR_INT
	ReductionAdd::reduce(tabSM, ptrNbDarUnderGM);
#endif

#ifdef DAR_LONG
	Reduction::reduce(add, addAtomic, tabSM, ptrNbDarUnderGM);
#endif
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__
void reductionIntraThread(hiprandState* tabGeneratorGM , entier* tabSM , entier nbDarByThread , float h)
    {
    const int TID = Thread1D::tid();
    const int TID_LOCAL = Thread1D::tidLocal();

    // Global Memory -> Register (optimization)
    hiprandState generatorThread = tabGeneratorGM[TID];

    entier nbDarUnderCurveThread = 0;

    float xAlea01;
    float yAlea0h;

    for (entier i = 1; i <= nbDarByThread; ++i)
	{
	xAlea01 = hiprand_uniform(&generatorThread); // in [0,1[
	yAlea0h = hiprand_uniform(&generatorThread) * h; // in [0,h[
	float thrownY = f(xAlea01);
	if (thrownY > yAlea0h)
	    {
	    nbDarUnderCurveThread++;
	    }

	}

    // stocker resultat du thread dans tabSM
    tabSM[TID_LOCAL] = nbDarUnderCurveThread;

    //Register -> Global Memory
    //Necessaire si on veut utiliser notre generator
    //	- dans d'autre kernel
    //	- avec d'autres nombres aleatoires !
    tabGeneratorGM[TID] = generatorThread;
    }

__device__ float f(float x)
    {
    return 4.f / (1.f + x * x);
    }

/*-------------------*\
|* reduce operator   *|
 \*------------------*/

__device__ long add(long x , long y)
    {
    return x + y;
    }

/**
 * Utiliser la methode system : atomicAdd(pointeurDestination, valeurSource)
 * ou la technique du lock si atomicAdd existe pas pour les long
 */
__device__ int volatile mutex = 0;	//variable global
__device__ void addAtomic(long* ptrX , long y)
    {
    Lock locker(&mutex);
    locker.lock();
    *ptrX += y;
    locker.unlock();
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

