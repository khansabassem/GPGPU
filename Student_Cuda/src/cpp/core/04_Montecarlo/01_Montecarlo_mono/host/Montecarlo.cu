#include "hip/hip_runtime.h"
#include "Montecarlo.h"

#include <iostream>
#include <assert.h>
#include <typeinfo>
#include <math.h>

#include "GM.h"
#include "Hardware.h"

using std::cout;
using std::endl;
using std::string;
using std::to_string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void createGenerator(hiprandState* tabGeneratorGM,int deviceId);

extern __global__ void montecarlo(hiprandState* tabGeneratorGM, entier nbDarByThread, entier* ptrNbDarUnderGM, float h);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

Montecarlo::Montecarlo(const Grid& grid , entier nbDarTotalAsk , double* ptrPiHat , float h , bool isVerbose) :
	RunnableGPU(grid, title(nbDarTotalAsk, h), isVerbose), // classe parente
	//
	nbDarTotalAsk(nbDarTotalAsk), //
	ptrPiHat(ptrPiHat), //
	h(h) //

    {
    const entier NB_THREAD = grid.threadCounts(); //directement en entier pour éviter un cast

    // Math : Calculer le nombre de flechette effectivment tirees!
	{
	this->nbDarByThread = nbDarTotalAsk / NB_THREAD;
	this->nbDarTotalEffective = nbDarByThread * NB_THREAD;// TODO Montecarlo

	assert(nbDarTotalAsk >= grid.threadCounts());
	assert(nbDarByThread > 0); // si =0, trop de threads et pas assez de dar!
	}

    // MM
	{
	this->sizeNbDarUnderGM = sizeof(entier); // [octet]

	// Montecarlo MM NbDarUnderGM (pas oublier de mettre a zero, avec un malloc0 par exemple)
	GM::malloc0(&ptrNbDarUnderGM, sizeNbDarUnderGM);

	this->sizeSM = sizeof(entier) * grid.threadByBlock(); // [octet]
	}

    // init : lancer le kernel de creation des generators
	{
	size_t sizeTabDevGeneratorGM = NB_THREAD * sizeof(hiprandState);

	// TODO Montecarlo MM pou  tabDevGeneratorGM
	 GM::malloc(&tabDevGeneratorGM, sizeTabDevGeneratorGM);
	// TODO Montecarlo lancer le kernel createGenerator
	int deviceId = Hardware::getDeviceId();
	createGenerator<<<dg, db>>>(tabDevGeneratorGM, deviceId);

	}
    }

Montecarlo::~Montecarlo(void)
    {
//MM (device free)
	{
	GM::free(tabDevGeneratorGM);
	GM::free(ptrNbDarUnderGM);
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

/**
 * Lancer kernel
 * Recuperer resultat coter host
 * Finaliser le calcul de pi
 */
void Montecarlo::run()
    {
    montecarlo<<<dg, db, sizeSM>>>(tabDevGeneratorGM , nbDarByThread , ptrNbDarUnderGM , h);
    GM::memcpyDToH(&nbDarUnderCurve, ptrNbDarUnderGM, sizeNbDarUnderGM);
    *ptrPiHat = nbDarUnderCurve/(double)nbDarTotalEffective * h; /// (double)nbDarTotalEffective;
    }

/*----------------*\
 |*	get        *|
 \*---------------*/

entier Montecarlo::getNbDarTotalEffective()
    {
    return nbDarTotalEffective;
    }

entier Montecarlo::getNbDarUnderCurve()
    {
    return nbDarUnderCurve;
    }

double Montecarlo::getInputGO()
    {
    return (nbDarTotalEffective / (double)1024 / (double)1024 / (double)1024) * sizeof(float) * 2;
    }

/*----------------*\
 |*  private       *|
 \*---------------*/

/**
 * static
 */
string Montecarlo::title(entier nbDarTotalAsk , float h)
    {
    return "Montecarlo-" + entierToString() + "-" + to_string(nbDarTotalAsk) + "-h" + to_string((int)h);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
