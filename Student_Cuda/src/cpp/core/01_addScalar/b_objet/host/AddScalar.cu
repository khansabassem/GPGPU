#include "hip/hip_runtime.h"
#include "AddScalar.h"

#include <iostream>
#include <assert.h>
#include "GM.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void addScalar(float a, float b, float* ptrDevSum);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

AddScalar::AddScalar(float a , float b , float* ptrSum) :
	a(a),//
	b(b), //
	ptrSum(ptrSum)
    {
    this->sizeFloat = sizeof(float); // [octet]

    // MM
	{
	GM::malloc(&prtDevSum, sizeFloat);
	}
    }

AddScalar::~AddScalar(void)
    {
    //MM
	{
	GM::free(prtDevSum);
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void AddScalar::run()
    {
    // Grid : specifier le nombre de thread
    dim3 dg(1, 1, 1);
    dim3 db(1, 1, 1);
    assert(dg.x * dg.y * dg.z * db.x * db.y * db.z == 1);// 1 seul thread suffit

    addScalar<<<dg,db>>>(a, b, prtDevSum); // assynchrone

    //Device::synchronize(); // necessaire only pour printf coter device

    // MM (Device -> Host)
	{
	GM::memcpyDToH(ptrSum, prtDevSum, sizeFloat); // MM = barriere synchronisation implicite
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
