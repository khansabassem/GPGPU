#include "hip/hip_runtime.h"
// Attention : Extension .cu

#include <iostream>
#include <stdio.h>
#include "cudas.h"
#include "GM.h"
#include "Kernel.h"

using std::cout;
using std::endl;

/**
 * http://www.icl.utes3/docs/cuda.htmk.edu/~mgatl
 */

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ static void addScalar(int a , int b , int* ptrDevSum);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Host			*|
 \*-------------------------------------*/

__host__ void addScalar() // __host__ facultatif
    {
    int a = 10;
    int b = 1;
    int sum;

    int* ptrSum = &sum;	// on host (CPU)
    int* ptrDevSum; 	// on device (GPU)

    // MM (memory managment)
    size_t size = sizeof(int); // [octet]
    GM::malloc(&ptrDevSum, size); // Device memory allocation (*)

    // Grid : Specifier number thread : ici 1 thread au total !
    dim3 dg(1, 1, 1);
    dim3 db = dim3(1, 1, 1); // syntaxe alternative

    addScalar<<<dg,db>>>(a,b,ptrDevSum); // asynchrone, call le kernel GPU addScalar

    Kernel::synchronize(); // necessaire only pour printf sur device

    // MM (memory management)
	{
	GM::memcpyDToH(ptrSum, ptrDevSum, size);   // Device -> Host, MM = barrier de synchronisation
	GM::free(ptrDevSum); //  free memory create in (*)
	}

    cout << "\n[Hello : Host   side : addScalar] " << a << " + " << b << " = " << sum << endl;
    }

/*--------------------------------------*\
 |*		Device			*|
 \*-------------------------------------*/

/**
 * Hyp : 1 seul thread, ie dg(1,1,1) et db (1,1,1)
 */
__global__ void addScalar(int a , int b , int* ptrDevSum)
    {
    *ptrDevSum = a + b; // Hyp: 1 seul thread (pas besoin de plus, pour additioner 2 nombre)

    // debug
    printf("[Hello : Device side : addScalar] %d + %d = %d", a, b, *ptrDevSum); // requiert  Kernel::synchronize() in host side after kernel
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

