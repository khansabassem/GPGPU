#include "hip/hip_runtime.h"
#include "Thread2D.h"
#include "Thread1D.h"
#include "cudas.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/**
 * output : void required !!
 * pattern entrelacement
 */
// ptrDev pointeur device, 2 espaces mémoire différents => ne pas les mélanger
__global__ void addVector(float* ptrDevV1 , float* ptrDevV2 , float* ptrDevW , int n)
    {
    const int NB_THREAD = Thread2D::nbThread();
    const int TID = Thread2D::tid();
    int s = TID;
    while (s < n)
	{
	ptrDevW[s] = ptrDevV1[s] + ptrDevV2[s];
	s += NB_THREAD;
	}

    //TODO AddVector pattern entrelacement
    }

/**
 * pattern 1<-->1
 * hyp: #thread=#caseVecteur=n => 10 case en 10 thread
 */
__global__ void addVector11(float* ptrDevV1 , float* ptrDevV2 , float* ptrDevW , int n)
    {
    const int TID = Thread2D::tid();
    ptrDevW[TID] = ptrDevV1[TID] + ptrDevV2[TID];
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

