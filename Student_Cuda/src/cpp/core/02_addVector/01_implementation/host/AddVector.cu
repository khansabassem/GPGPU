#include "hip/hip_runtime.h"
#include "AddVector.h"

#include <iostream>
#include <assert.h>

#include "Kernel.h"
#include "GM.h"
#include "Bandwidth.h"
#include "VectorTools.h"

using std::cout;
using std::endl;
using std::to_string;
using std::string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void addVector(float* ptrDevV1, float* ptrDevV2, float* ptrDevW,int n);
extern __global__ void addVector11(float* ptrDevV1, float* ptrDevV2, float* ptrDevW, int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

AddVector::AddVector(const Grid& grid , float* ptrV1 , float* ptrV2 , float* ptrW , int n , Pattern pattern,bool isVerbose) :
	RunnableGPU(grid, title() + "-" + toString(pattern) + "-" + to_string(n),isVerbose), // classe parente
//
	ptrV1(ptrV1), //
	ptrV2(ptrV2), //
	ptrW(ptrW), //
	n(n), //
	pattern(pattern)

    {
    this->sizeVector = sizeof(float)*n; // TODO AddVector // octet

    // MM (malloc Device)
	{
	GM::malloc(&ptrDevV1, sizeVector); //gm pour global memory
	GM::malloc(&ptrDevV2, sizeVector);
	GM::malloc(&ptrDevW, sizeVector);

	}

    }

AddVector::~AddVector(void)
    {
    //MM (device free)
	{
	GM::free(ptrDevV1);
	GM::free(ptrDevV2);
	GM::free(ptrDevW);
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

/**
 * override
 */
void AddVector::run()
    {
    // MM (copy Host->Device)
	{
	Bandwidth bandwidth(sizeVector * 2, "[" + title() + "] : Host -> GM :");

	GM::memcpyHToD(ptrDevV1, ptrV1, sizeVector);
	GM::memcpyHToD(ptrDevV2, ptrV2, sizeVector);
	// TODO AddVector

	if (isVerbose) // dans Runable ou RunnableGPU
	    {
	    cout << bandwidth << endl;
	    }
	}

    switch (pattern)
	{
    case ENTRELACEMNT:
	{
	addVector<<<dg,db>>>(ptrDevV1, ptrDevV2, ptrDevW, n);
	break;
	}

    case UN_UN:
	{
	assert(db.x * db.y * db.z * dg.x * dg.y * dg.z == n);
	addVector11<<<dg,db>>>(ptrDevV1, ptrDevV2, ptrDevW, n);
	break;
	}

    default:
	{
	assert(false);
	break;
	}
	}

// MM (Device -> Host)
	{
	Bandwidth bandwidth(sizeVector, "[" + title() + "] : GM -> Host :");
	//barriere de memo implicite
	GM::memcpyDToH(ptrW, ptrDevW, sizeVector); // ce qui est à droite va à gauche
	// on ne sait pas si le kernel a terminé le travail.

	if (isVerbose)
	    {
	    cout << bandwidth << endl; // best with DMA in VectorTools
	    }
	}
    }

/**
 * override
 */
double AddVector::getInputGO()
    {
    return ((long)2 * (long)n * (long)sizeof(float)) / (double)((long)1024 * (long)1024 * (long)1024);
    }

/**
 * override
 */
double AddVector::getOutputGO()
    {
    return ((long)1 * (long)n * (long)sizeof(float)) / (double)((long)1024 * (long)1024 * (long)1024);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

string AddVector::title()
    {
    if (VectorTools::isDMA())
	{
	return "Addvector-DMA";
	}
    else
	{
	return "Addvector";
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
