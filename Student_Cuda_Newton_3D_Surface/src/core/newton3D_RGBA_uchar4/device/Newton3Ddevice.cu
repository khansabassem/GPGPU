#include "hip/hip_runtime.h"
#include "Newton3DMath.h"
#include "Thread2D.h"
#include "Indices_GPU.h"
#include "DomainMath3D_GPU.h"
#include "cudas.h"


using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void newton3DCuda(float3* tabVerticesXYZGM , uchar4* tabVerticesColorGM , int w , int h , DomainMath3D domaineMath , int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * w nbPoint en x
 * h nbPoint en y
 */
__global__ void newton3DCuda(float3* tabVerticesXYZGM , uchar4* tabVerticesColorGM , int w , int h , DomainMath3D domaineMath , int n)
    {
    float epsilon =0.1f*powf(0.90f,n);// 0.1f;

    Newton3DMath_RGBA newtonMath(epsilon, n);

    const int TID = Thread2D::tid();
    const int NB_THREAD = Thread2D::nbThread();

    const int WH = w * h;

//    uchar4 colorxy;// by me
    uchar4 color;
    float3 sommet;

    float x;
    float y;
    float DX;
    float DY;

    int vertexI;
    int vertexJ;

    domaineMath.delta(w, h, &DX, &DY);

    int s = TID;
    float hue, sat, val;

    while (s < WH)
	{
	Indices::toIJ(s, w, &vertexI, &vertexJ); // update (vertexI, vertexJ)

	domaineMath.toXY(DX, DY, vertexI, vertexJ, &x, &y);

	newtonMath.colorXY(&color, x, y);//couleur en format RVB
	newtonMath.RGBtoHUE( color.x, color.y, color.z, &hue, &sat, &val);
//	printf("hue : %f \n sat: %f\n, val: %f\n", hue, sat, val);
	newtonMath.sommetXY(&sommet, hue, sat, val, x, y);//RVB -> HSB,  puis extraire la valeur de sommet selon the Hue de HSB.


	tabVerticesColorGM[s] = color;
	tabVerticesXYZGM[s] = sommet;

	s += NB_THREAD;
	}

    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

