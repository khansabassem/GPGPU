#include "hip/hip_runtime.h"
#include "Newton3D.h"

#include <assert.h>


#include "Maths.h"
#include "cudas.h"

using std::to_string;

using gpu::DomainMath3D;
using gpu::SurfaceStrip_RGBA_uchar4;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void newton3DCuda(float3* tabVerticesXYZGM, uchar4* tabVerticesColorGM, int w, int h, DomainMath3D domaineMath, int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

//Newton3D_RGBA::Newton3D_RGBA(const Grid& grid , int w , int h , const DomainMath3D& domaineMath , float dt , int n) :
Newton3D_RGBA::Newton3D_RGBA(const Grid& grid , uint w , uint h , uint nMin, uint nMax, const DomainMath3D& domaineMath) :
	// Parent
	SurfaceStrip_RGBA_uchar4(w, h, domaineMath), //
	// Inputs
	n(n), //
	dg(grid.dg), //
	db(grid.db), //,
	w(w),
	h(h),
	// Tools
//	variateurAnimation(cpu::Interval<float>(0, 2 * PI), dt)
	variateurN(gpu::Interval<int>(nMin, nMax), 1) //
    {
    // Tools
    this->title = "Newton3D_RGBA_uchar4";
    this->t = 0;
    }

Newton3D_RGBA::~Newton3D_RGBA()
    {
    // rien
    }

/*-------------------------*\
 |*	Surcharge	   *|
 \*------------------------*/

/**
 * Override
 * Call periodicly by the api
 */
void Newton3D_RGBA::fillVertex(float3* tabVerticesXYZGM , uchar4* tabVerticesColorGM , unsigned int w , unsigned int h , const gpu::DomainMath3D& domaineMath)
    {
    newton3DCuda<<<dg,db>>>(tabVerticesXYZGM,tabVerticesColorGM,w,h,domaineMath,n);
}

/**
 * Override
 * Call periodicly by the api
 */
void Newton3D_RGBA::animationStep(bool& isNeedUpdate)
{
//this->t = variateurAnimation.varierAndGet(); // in [0,2pi]
    this->n = variateurN.varierAndGet(); // in [0,2pi]

}

/**
 * Override
 * Call periodicly by the api
 */
void Newton3D_RGBA::paintPrimitives(Graphic2D& graphic2D)
{
graphic2D.setFont(TIMES_ROMAN_24);

float r = 0.5f;
float g = 0.5f;
float b = 0.5f;

graphic2D.setColorRGB(r, g, b);

// TOP : Para Animation
    {
    string message = "t = " + to_string(t);
    graphic2D.drawTitleTop(message);
    }

// BOTTOM : Title
    {
    graphic2D.drawTitleBottom(title);
    }
}

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

