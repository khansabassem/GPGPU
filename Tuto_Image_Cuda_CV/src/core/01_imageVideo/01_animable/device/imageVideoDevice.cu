#include "hip/hip_runtime.h"
#include "Thread2D.h"
#include "cudas.h"

#include "ImageVideoMath.h"

#include "Indices_GPU.h"
using namespace gpu;

// Attention : 	Choix du nom est impotant!
//		VagueDevice.cu et non Vague.cu
// 		Dans ce dernier cas, probl�me de linkage, car le nom du .cu est le meme que le nom d'un .cpp (host)
//		On a donc ajouter Device (ou n'importequoi) pour que les noms soient diff�rents!

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void toGray(uchar4* tabPixelsGM , uint w , uint h , float t);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void toGray(uchar4* tabPixelsGM , uint w , uint h , float t)
    {
    ImageVideoMath math(w, h);

    const int TID = Thread2D::tid();
    const int NB_THREAD = Thread2D::nbThread();
    const int WH = w * h;

    int pixelI;	// in [0,h[
    int pixelJ; // in [0,w[

    int s = TID;
    while (s < WH)
	{
	Indices::toIJ(s, w, &pixelI, &pixelJ); 	// update (pixelI, pixelJ)

	math.colorIJ(&tabPixelsGM[s], pixelI, pixelJ, t); 	//update color  // ici pas besoin de i, j,t, optimisation possible

	s += NB_THREAD;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

