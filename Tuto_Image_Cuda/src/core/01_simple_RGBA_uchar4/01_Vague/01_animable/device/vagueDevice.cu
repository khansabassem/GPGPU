#include "hip/hip_runtime.h"
#include "Thread2D.h"
#include "cudas.h"

#include "VagueMath.h"

#include "Indices_GPU.h"
using namespace gpu;

// Attention : 	Choix du nom est impotant!
//		VagueDevice.cu et non Vague.cu
// 		Dans ce dernier cas, probl�me de linkage, car le nom du .cu est le meme que le nom d'un .cpp (host)
//		On a donc ajouter Device (ou n'importequoi) pour que les noms soient diff�rents!

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void vague(uchar4* tabPixelsGM , uint w , uint h , int t);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void vague(uchar4* tabPixelsGM , uint w , uint h , int t)
    {
    VagueMath vagueMath(w, h,t);

    const int WH = w * h;
    const int TID = Thread2D::tid();
    const int NB_THREAD = Thread2D::nbThread();

    int i;	// in [0,h[
    int j; 	// in [0,w[

    int s = TID;  // in [0,...
    while (s < WH)
	{
	Indices::toIJ(s, w, &i, &j); 	// update (i, j)

	vagueMath.colorIJ(&tabPixelsGM[s], i, j); 	// update ptrDevPixels[s]

	s += NB_THREAD;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

