#include "hip/hip_runtime.h"
#include "Vague.h"

#include <iostream>
#include <assert.h>

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void vague(uchar4* tabPixelsGM,uint w, uint h,int t);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Vague::Vague(const Grid& grid , uint w , uint h , int dt,bool isVerbose) :
	Animable_I<uchar4>(grid, w, h, "Vague-Cuda-RGBA-uchar4",isVerbose),//
	dt(dt)
    {
    // Tools
    this->t = 0; // protected dans Animable
    }

Vague::~Vague()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void Vague::process(uchar4* tabPixelsGM , uint w , uint h , const DomaineMath& domaineMath)
    {
vague<<<dg,db>>>(tabPixelsGM,w,h,t);
}

/**
 * Override
 * Call periodicly by the API
 */
void Vague::animationStep()
{
t += dt;
}

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

