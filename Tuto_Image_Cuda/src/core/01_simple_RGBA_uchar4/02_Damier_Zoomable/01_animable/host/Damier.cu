#include "hip/hip_runtime.h"
#include "Damier.h"

#include <iostream>
#include <assert.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void damier(uchar4* tabPixelsGM , uint w , uint h , DomaineMath domaineMath , uint n , float t);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Damier::Damier(const Grid& grid , uint w , uint h , float dt , uint n , const DomaineMath& domaineMath,bool isVerbose) :
	Animable_I<uchar4>(grid, w, h, "Damier-Cuda-RGBA-uchar4", domaineMath,isVerbose), variateurAnimation(Interval<float>(0, 2 * PI), dt), //
	n(n)
    {
    // Tools
    this->t = 0;  // protected dans Animable
    }

Damier::~Damier()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 */
void Damier::process(uchar4* tabPixelsGM , uint w , uint h , const DomaineMath& domaineMath)
    {
damier<<<dg,db>>>(tabPixelsGM,w,h,domaineMath,n,t);
}

/**
 * Override
 * Call periodicly by the API
 */
void Damier::animationStep()
{
this->t = variateurAnimation.varierAndGet(); // in [0,2pi]
}

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

