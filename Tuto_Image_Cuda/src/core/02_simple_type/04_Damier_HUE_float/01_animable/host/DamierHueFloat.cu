#include "hip/hip_runtime.h"
#include "DamierHueFloat.h"

#include <iostream>
#include <assert.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void damierHueFloat(float* tabPixelsGM , uint w , uint h , DomaineMath domaineMath , uint n , float t);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

DamierHueFloat::DamierHueFloat(const Grid& grid , uint w , uint h , float dt , uint n , const DomaineMath& domaineMath,bool isVerbose) :
	Animable_I<float>(grid, w, h, "Damier-Cuda-Hue-float", domaineMath,isVerbose), //
	variateurAnimation(Interval<float>(0, 2 * PI), dt), //
	n(n)
    {
    // Tools
    this->t = 0;  // protected dans Animable
    }

DamierHueFloat::~DamierHueFloat()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 */
void DamierHueFloat::process(float* tabPixelsGM , uint w , uint h , const DomaineMath& domaineMath)
    {
damierHueFloat<<<dg,db>>>(tabPixelsGM,w,h,domaineMath,n,t);
}

/**
 * Override
 * Call periodicly by the API
 */
void DamierHueFloat::animationStep()
{
this->t = variateurAnimation.varierAndGet(); // in [0,2pi]
}

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

