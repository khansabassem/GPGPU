#include "hip/hip_runtime.h"
#include "DamierHSBAFloat.h"

#include <iostream>
#include <assert.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void damierHSBAFloat(float4* tabPixelsGM , uint w , uint h , DomaineMath domaineMath , uint n , float t);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

DamierHSBAFloat::DamierHSBAFloat(const Grid& grid , uint w , uint h , float dt , uint n , const DomaineMath& domaineMath,bool isVerbose) :
	Animable_I<float4>(grid, w, h, "Damier-Cuda-HSBA-float4", domaineMath,isVerbose), //
	variateurAnimation(Interval<float>(0, 2 * PI), dt), //
	n(n)
    {
    // Tools
    this->t = 0;  // protected dans Animable
    }

DamierHSBAFloat::~DamierHSBAFloat()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 */
void DamierHSBAFloat::process(float4* tabPixelsGM , uint w , uint h , const DomaineMath& domaineMath)
    {
damierHSBAFloat<<<dg,db>>>(tabPixelsGM,w,h,domaineMath,n,t);
}

/**
 * Override
 * Call periodicly by the API
 */
void DamierHSBAFloat::animationStep()
{
this->t = variateurAnimation.varierAndGet(); // in [0,2pi]
}

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

