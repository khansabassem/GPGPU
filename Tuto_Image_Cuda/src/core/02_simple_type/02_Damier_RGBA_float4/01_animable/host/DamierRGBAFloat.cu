#include "hip/hip_runtime.h"
#include "DamierRGBAFloat.h"

#include <iostream>
#include <assert.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void damierRGBAFloat(float4* tabPixelsGM , uint w , uint h , DomaineMath domaineMath , uint n , float t);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

DamierRGBAFloat::DamierRGBAFloat(const Grid& grid , uint w , uint h , float dt , uint n , const DomaineMath& domaineMath,bool isVerbose) :
	Animable_I<float4>(grid, w, h, "Damier-Cuda-RGBA-float4", domaineMath,isVerbose), //
	variateurAnimation(Interval<float>(0, 2 * PI), dt), //
	n(n)
    {
    // Inputs
    this->n = n;

    // Tools
    this->t = 0;  // protected dans Animable
    }

DamierRGBAFloat::~DamierRGBAFloat()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 */
void DamierRGBAFloat::process(float4* tabPixelsGM , uint w , uint h , const DomaineMath& domaineMath)
    {
damierRGBAFloat<<<dg,db>>>(tabPixelsGM,w,h,domaineMath,n,t);
}

/**
 * Override
 * Call periodicly by the API
 */
void DamierRGBAFloat::animationStep()
{
this->t = variateurAnimation.varierAndGet(); // in [0,2pi]
}

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

