#include "hip/hip_runtime.h"
#include "Thread2D.h"
#include "cudas.h"

#include "DamierRGBAFloatMath.h"

#include "Indices_GPU.h"
#include "DomaineMath_GPU.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void damierRGBAFloat(float4* tabPixelsGM , uint w , uint h , DomaineMath domaineMath , uint n , float t);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void damierRGBAFloat(float4* tabPixelsGM , uint w , uint h , DomaineMath domaineMath , uint n , float t)
    {
    DamierRGBAFloatMath damierRGBAFloatMath(n,t);

    const int TID = Thread2D::tid();
    const int NB_THREAD = Thread2D::nbThread();
    const int WH = w * h;

    double x;
    double y;

    int i; // in [0,h[
    int j; // in [0,w[

    int s = TID;
    while (s < WH)
	{
	Indices::toIJ(s, w, &i, &j); // update (i, j)

	// (i,j) domaine ecran
	// (x,y) domaine math
	domaineMath.toXY(i, j, &x, &y); //  (i,j) -> (x,y)

	damierRGBAFloatMath.colorXY(&tabPixelsGM[s], x, y); // update ptrDevPixels[s]

	s += NB_THREAD;
	}

    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

