#include "hip/hip_runtime.h"
#include "Thread2D.h"
#include "cudas.h"
#include "VagueGrayMath.h"

#include "Indices_GPU.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void vagueGray(uchar* tabPixelsGM , uint w , uint h , int t);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void vagueGray(uchar* tabPixelsGM , uint w , uint h , int t)
    {
    VagueGrayMath vagueGrayMath(w, h,t);

    const int TID = Thread2D::tid();
    const int NB_THREAD = Thread2D::nbThread();
    const int WH = w * h;

    int i; // in [0,h[
    int j; // in [0,w[

    int s = TID;
    while (s < WH)
	{
	Indices::toIJ(s, w, &i, &j); 	// update (i, j)

	vagueGrayMath.colorIJ(&tabPixelsGM[s], i, j); 	// update ptrDevPixels[s]

	s += NB_THREAD;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

