#include "hip/hip_runtime.h"
#include "VagueGray.h"

#include <iostream>
#include <assert.h>

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void vagueGray(uchar* tabPixelsGM,uint w, uint h,int t);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

VagueGray::VagueGray(const Grid& grid , uint w , uint h , int dt,bool isVerbose) :
	Animable_I<uchar>(grid, w, h, "Vague-Cuda-GRAY-uchar",isVerbose), //
	dt(dt)
    {
    // Tools
    this->t = 0; // protected dans Animable
    }

VagueGray::~VagueGray()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void VagueGray::process(uchar* tabPixelsGM , uint w , uint h , const DomaineMath& domaineMath)
    {
vagueGray<<<dg,db>>>(tabPixelsGM,w,h,t);
}

/**
 * Override
 * Call periodicly by the API
 */
void VagueGray::animationStep()
{
t += dt;
}

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

