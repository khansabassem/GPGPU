#include "hip/hip_runtime.h"
#include "ReductionAddIntI.h"

#include "GM.h"
#include "Grid.h"

#include <iostream>
#include <assert.h>

using std::cout;
using std::endl;
using std::to_string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void KAddIntProtocoleI(int* ptrSumGM);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

ReductionAddIntI::ReductionAddIntI(const Grid& grid , int* ptrSum , bool isVerbose) :
	RunnableGPU(grid, "ReductionAddIntI-" + to_string(grid.threadCounts()), isVerbose), // classe parente
	ptrSum(ptrSum)
    {
    // Memory mgmt GM
    GM::mallocInt(&ptrSumGM, 0); // % car le pointeur ne valait rien avant, ca valeur va changer
    this->sizeSM = grid.threadByBlock() * sizeof(int);
    }

ReductionAddIntI::~ReductionAddIntI()
    {
    GM::free(ptrSumGM);
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void ReductionAddIntI::run()
    {
    // TODO ReductionAddIntI
    KAddIntProtocoleI<<<dg,db,sizeSM>>>(ptrSumGM);
    GM::memcpyDToH_int(ptrSum, ptrSumGM);
    // appeler le kernel
    // recuperer le resulat coter host
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
