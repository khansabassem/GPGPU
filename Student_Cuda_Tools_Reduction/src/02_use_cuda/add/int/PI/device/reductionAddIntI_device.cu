#include "hip/hip_runtime.h"
#include "Thread1D.h"
#include "cudas.h"

#include "ReductionAdd.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void reductionIntraThread(int* tabSM);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

__global__ void KAddIntProtocoleI(int* ptrSumGM)
    {
    // TODO ReductionAddIntI

    // Declaration tabSM
    // ReductionIntraThread
    // ReductionAdd
    extern __shared__ int tabSM[];
    reductionIntraThread (tabSM);
    __syncthreads();
    ReductionAdd::reduce(tabSM, ptrSumGM);
    //syncthreads implicite
    // __syncthreads(); // des threads de meme block!// Question : utile? ou?
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * 1 partout en tabSM
 */
__device__ void reductionIntraThread(int* tabSM)
    {
    const int TID_LOCAL = Thread1D::tidLocal();
    tabSM[TID_LOCAL] = 1; // global pour le PII

    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

