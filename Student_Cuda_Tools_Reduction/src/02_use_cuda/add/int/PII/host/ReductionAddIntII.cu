#include "hip/hip_runtime.h"
#include "ReductionAddIntII.h"

#include "GM.h"
#include "Grid.h"

#include <iostream>
#include <assert.h>

using std::cout;
using std::endl;
using std::to_string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void KAddIntProtocoleII(int* ptrSumGM);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

ReductionAddIntII::ReductionAddIntII(const Grid& grid , int* ptrSum , bool isVerbose) :
	RunnableGPU(grid, "ReductionAddIntII-" + to_string(grid.threadCounts()), isVerbose), // classe parente
	ptrSum(ptrSum)
    {
    GM::mallocInt(&ptrSumGM, 0); // % car le pointeur ne valait rien avant, ca valeur va changer
    this->sizeSM = grid.threadByBlock() * sizeof(int);
    }

ReductionAddIntII::~ReductionAddIntII()
    {
    GM::free(ptrSumGM);
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void ReductionAddIntII::run()
    {
    KAddIntProtocoleII<<<dg,db,sizeSM>>>(ptrSumGM);
    GM::memcpyDToH_int(ptrSum, ptrSumGM);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
