#include "hip/hip_runtime.h"
#include "Thread1D.h"
#include "cudas.h"

#include "ReductionAdd.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void reductionIntraThread(int* tabSM);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

__global__ void KAddIntProtocoleII(int* ptrSumGM)
    {
    extern __shared__ int tabSM[];
    reductionIntraThread (tabSM);
    __syncthreads();
    ReductionAdd::reduce(tabSM, ptrSumGM);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * TID partout en tabSM
 */
__device__ void reductionIntraThread(int* tabSM)
    {
    const int TID = Thread1D::tid();
    tabSM[threadIdx.x] = TID; // global pour le PII
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

