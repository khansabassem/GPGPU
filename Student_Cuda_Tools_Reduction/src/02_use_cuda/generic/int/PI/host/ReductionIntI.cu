#include "hip/hip_runtime.h"
#include "ReductionIntI.h"

#include "GM.h"
#include "Grid.h"

#include <iostream>
#include <assert.h>

using std::cout;
using std::endl;
using std::to_string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void KIntProtocoleI(int* ptrSumGM);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

ReductionIntI::ReductionIntI(const Grid& grid , int* ptrSum , bool isVerbose) :
	RunnableGPU(grid, "ReductionIntI-" + to_string(grid.threadCounts()), isVerbose), // classe parente
	ptrSum(ptrSum)
    {
    GM::mallocInt(&ptrSumGM, 0); // % car le pointeur ne valait rien avant, ca valeur va changer
    this->sizeSM = grid.threadByBlock() * sizeof(int);
    }

ReductionIntI::~ReductionIntI()
    {
    GM::free(ptrSumGM);
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void ReductionIntI::run()
    {
    KIntProtocoleI<<<dg,db,sizeSM>>>(ptrSumGM);
    GM::memcpyDToH_int(ptrSum, ptrSumGM);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
