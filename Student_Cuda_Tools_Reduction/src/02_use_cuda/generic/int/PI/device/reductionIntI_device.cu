#include "hip/hip_runtime.h"
#include "Thread1D.h"
#include "cudas.h"

#include "Reduction.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void reductionIntraThread(int* tabSM);

// Operateur reduction
static __device__ int add(int x , int y);
static __device__ void addAtomicV1(int* ptrX , int y);
static __device__ void addAtomicV2(int* ptrX , int y);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

__global__ void KIntProtocoleI(int* ptrSumGM)
    {
    extern __shared__ int tabSM[];
    reductionIntraThread (tabSM);
    __syncthreads();

    Reduction::reduce(add, addAtomicV1, tabSM, ptrSumGM);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * 1 partout en tabSM
 */
__device__ void reductionIntraThread(int* tabSM)
    {
    tabSM[threadIdx.x] = 1;
    }

/*----------------------------*\
|*	Operateur reduction    *|
 \*---------------------------*/

__device__ int add(int x , int y)
    {
    return x + y;
    }

/**
 * Utiliser la methode system
 *
 * 		atomicAdd(pointeurDestination, valeurSource);
 */
__device__ void addAtomicV1(int* ptrX , int y)
    {
    atomicAdd(ptrX, y);
    }

/**
 * 10x plus lent,mais plus flexible!
 * Necessaire si un vrai operateur atomic n'existe pas
 * Necessaire aussi pour des objets par exemple
 */
__device__ int volatile mutex = 0;	//variable global
__device__ void addAtomicV2(int* ptrX , int y)
    {
    Lock locker(&mutex);
    locker.lock();
    (*ptrX) += y;
    locker.unlock();
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

