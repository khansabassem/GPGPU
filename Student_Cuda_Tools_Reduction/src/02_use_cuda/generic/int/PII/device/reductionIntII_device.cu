#include "hip/hip_runtime.h"
#include "Thread1D.h"
#include "cudas.h"

#include "Reduction.h"
#include "Lock.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void reductionIntraThread(int* tabSM);

// Operateur reduction
static __device__ int add(int x , int y);
static __device__ void addAtomicV1(int* ptrX , int y);
static __device__ void addAtomicV2(int* ptrX , int y);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

__global__ void KIntProtocoleII(int* ptrSumGM)
    {
    extern __shared__ int tabSM[];
    reductionIntraThread (tabSM);
    __syncthreads();

    Reduction::reduce(add, addAtomicV1, tabSM, ptrSumGM);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * TID partout en tabSM
 */
__device__ void reductionIntraThread(int* tabSM)
    {
    const int TID = Thread1D::tid();
    tabSM[threadIdx.x] = TID; // global pour le PII
    }

/*----------------------------*\
|*	Operateur reduction    *|
 \*---------------------------*/

__device__ int add(int x , int y)
    {
    return x + y;
    }

/**
 * Utiliser la methode system : atomicAdd(pointeurDestination, valeurSource);
 */
__device__ void addAtomicV1(int* ptrX , int y)
    {
    atomicAdd(ptrX, y);
    }

/**
 * Une alternative, moins performante, mais generalisable serait d'employer un lock
 * Tip : le Lock est implementer avec deux methodes atomic
 */
__device__ int volatile mutex = 0;	//variable global
__device__ void addAtomicV2(int* ptrX , int y)
    {
    Lock locker(&mutex);
    locker.lock();

    (*ptrX) += y;

    locker.unlock();
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

