#include "hip/hip_runtime.h"
#include "ReductionIntII.h"

#include "GM.h"
#include "Grid.h"

#include <iostream>
#include <assert.h>

using std::cout;
using std::endl;
using std::to_string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void KIntProtocoleII(int* ptrSumGM);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

ReductionIntII::ReductionIntII(const Grid& grid , int* ptrSum , bool isVerbose) :
	RunnableGPU(grid, "ReductionIntII-" + to_string(grid.threadCounts()), isVerbose), // classe parente
	ptrSum(ptrSum)
    {
    GM::mallocInt(&ptrSumGM, 0); // % car le pointeur ne valait rien avant, ca valeur va changer
    this->sizeSM = grid.threadByBlock() * sizeof(int);
    }

ReductionIntII::~ReductionIntII()
    {
    GM::free(ptrSumGM);
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void ReductionIntII::run()
    {
    KIntProtocoleII<<<dg,db,sizeSM>>>(ptrSumGM);
    GM::memcpyDToH_int(ptrSum, ptrSumGM);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
