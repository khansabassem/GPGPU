#include "hip/hip_runtime.h"
#include "ReductionLongII.h"

#include "GM.h"
#include "Grid.h"

#include <iostream>
#include <assert.h>

using std::cout;
using std::endl;
using std::to_string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void KLongProtocoleII(long* ptrSumGM);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

ReductionLongII::ReductionLongII(const Grid& grid , long* ptrSum , bool isVerbose) :
	RunnableGPU(grid, "ReductionLongII-" + to_string(grid.threadCounts()), isVerbose), // classe parente
	ptrSum(ptrSum)
    {
    GM::mallocLong(&ptrSumGM, 0); // % car le pointeur ne valait rien avant, ca valeur va changer
    this->sizeSM = grid.threadByBlock() * sizeof(long);
    }

ReductionLongII::~ReductionLongII()
    {
    GM::free(ptrSumGM);
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void ReductionLongII::run()
    {
    KLongProtocoleII<<<dg,db,sizeSM>>>(ptrSumGM);
    GM::memcpyDToH_long(ptrSum, ptrSumGM);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
