#include "hip/hip_runtime.h"
#include "Thread1D_long.h"
#include "cudas.h"

#include "Reduction.h"
#include "Lock.h"

#include <stdio.h>

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void reductionIntraThread(long* tabSM);

// Operateur reduction
static __device__ long add(long x , long y);
static __device__ void addAtomic(long* ptrX , long y);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/**
 * TID partout en tabSM
 */
__global__ void KLongProtocoleII(long* ptrSumGM)
    {
    extern __shared__ long tabSM[];
    reductionIntraThread (tabSM);
    __syncthreads();

    Reduction::reduce(add, addAtomic, tabSM, ptrSumGM);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * TID partout en tabSM
 */
__device__ void reductionIntraThread(long* tabSM)
    {
    //		Juste (v1):	const long TID=((long)blockDim.x * (long) blockIdx.x)+ (long)threadIdx.x;
    //
    //		Juste (V2):	const long TID=Thread1D_long::tid();
    //
    //		Juste (V3):	const long TID=Thread2D_long::tid();
    const long TID = Thread1D_long::tid();
    tabSM[threadIdx.x] = TID; // global pour le PII
    }

/*----------------------------*\
|*	Operateur reduction    *|
 \*---------------------------*/

__device__ long add(long x , long y)
    {
    return x + y;
    }

/**
 * Utiliser la methode system, si elle existe
 *
 * 	atomicAdd(pointeurDestination, valeurSource)
 *
 * ou la technique du lock vu precedement!
 *
 * Question : atomicAdd pour les long existe?
 */
__device__ int volatile mutex = 0;	//variable global
__device__ void addAtomic(long* ptrX , long y)
    {
    Lock locker(&mutex);
    locker.lock();

    (*ptrX) += y;

    locker.unlock();
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

