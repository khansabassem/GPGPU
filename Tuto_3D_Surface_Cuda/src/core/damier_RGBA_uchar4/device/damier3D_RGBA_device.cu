#include "hip/hip_runtime.h"
#include "Damier3DMath_RGBA.h"
#include "Thread2D.h"
#include "Indices_GPU.h"
#include "DomainMath3D_GPU.h"
#include "cudas.h"


using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void damier3DCuda(float3* tabVerticesXYZGM , uchar4* tabVerticesColorGM , int w , int h , DomainMath3D domaineMath , int n , float t);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * w nbPoint en x
 * h nbPoint en y
 */
__global__ void damier3DCuda(float3* tabVerticesXYZGM , uchar4* tabVerticesColorGM , int w , int h , DomainMath3D domaineMath , int n , float t)
    {
    Damier3DMath_RGBA damierMath(n);

    const int TID = Thread2D::tid();
    const int NB_THREAD = Thread2D::nbThread();

    const int WH = w * h;

    uchar4 color;
    float3 sommet;

    float x;
    float y;
    float DX;
    float DY;

    int vertexI;
    int vertexJ;

    domaineMath.delta(w, h, &DX, &DY);

    int s = TID;
    while (s < WH)
	{
	Indices::toIJ(s, w, &vertexI, &vertexJ); // update (vertexI, vertexJ)

	domaineMath.toXY(DX, DY, vertexI, vertexJ, &x, &y);

	damierMath.sommetXY(&sommet, x, y, t); // update sommet
	damierMath.colorZ(&color, sommet.z); // update color

	tabVerticesColorGM[s] = color;
	tabVerticesXYZGM[s] = sommet;

	s += NB_THREAD;
	}

    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

