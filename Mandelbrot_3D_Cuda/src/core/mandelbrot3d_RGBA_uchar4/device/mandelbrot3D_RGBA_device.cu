#include "hip/hip_runtime.h"
#include "Mandelbrot3DMath_RGBA.h"
#include "Thread2D.h"
#include "Indices_GPU.h"
#include "DomainMath3D_GPU.h"
#include "cudas.h"


using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void mandelbrot3DCuda(float3* tabVerticesXYZGM , uchar4* tabVerticesColorGM , int w , int h , DomainMath3D domaineMath , int n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * w nbPoint en x
 * h nbPoint en y
 */
__global__ void mandelbrot3DCuda(float3* tabVerticesXYZGM , uchar4* tabVerticesColorGM , int w , int h , DomainMath3D domaineMath , int n)
    {
    Mandelbrot3DMath_RGBA mandelbrotMath(n);

    const int TID = Thread2D::tid();
    const int NB_THREAD = Thread2D::nbThread();

    const int WH = w * h;

    uchar4 color;
    float3 sommet;

    float x;
    float y;
    float DX;
    float DY;

    int vertexI;
    int vertexJ;



    domaineMath.delta(w, h, &DX, &DY);

    int s = TID;
    float hue;
    while (s < WH)
	{
	Indices::toIJ(s, w, &vertexI, &vertexJ); // update (vertexI, vertexJ)

	domaineMath.toXY(DX, DY, vertexI, vertexJ, &x, &y);


	mandelbrotMath.colorXY(&color , &hue, x , y);
	mandelbrotMath.sommetXY(&sommet, hue,x, y);

	tabVerticesColorGM[s] = color;
	tabVerticesXYZGM[s] = sommet;

	s += NB_THREAD;
	}

    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

