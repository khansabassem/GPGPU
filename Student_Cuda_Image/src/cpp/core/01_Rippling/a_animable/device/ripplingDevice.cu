#include "hip/hip_runtime.h"
#include "Thread2D.h"
#include "Thread1D.h"
#include "cudas.h"

#include "Indices_GPU.h"

#include "RipplingMath.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

__global__ void rippling(uchar4* tabPixelsGM , uint w , uint h , float t)
    {
    RipplingMath ripplingMath = RipplingMath(w , h, t);

    const int TID = Thread2D::tid();
    const int NB_THREAD = Thread2D::nbThread();
    const int WH = w * h;
    int pixelI, pixelJ;
    int s = TID;
    while(s<WH){

	Indices::toIJ(s, w, &pixelI, &pixelJ);

	ripplingMath.colorIJ(&tabPixelsGM[s], pixelI, pixelJ);

	s+= NB_THREAD;
        }
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

