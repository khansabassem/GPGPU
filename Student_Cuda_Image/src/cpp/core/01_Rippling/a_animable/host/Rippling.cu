#include "hip/hip_runtime.h"
#include "Rippling.h"

#include <iostream>
#include <assert.h>


#include <assert.h>

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void rippling(uchar4* tabPixelsGM,uint w, uint h,float t);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Rippling::Rippling(const Grid& grid, uint w, uint h, float dt,bool isVerbose) :
	Animable_I<uchar4>(grid, w, h, "Rippling-Cuda-uchar4",isVerbose)// super classe
    {
    assert(w == h); // specific rippling

    // Animation
    this->dt = dt;
    this->t = 0; // protected dans Animable
    }

Rippling::~Rippling()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car image pas zoomable
 */
void Rippling::process(uchar4* tabPixelsGM, uint w, uint h, const DomaineMath& domaineMath)
    {
    rippling<<<Animable_I::dg,Animable_I::db>>>(tabPixelsGM, w, h, t);
    }

/**
 * Override
 * Call periodicly by the API
 */
void Rippling::animationStep()
    {
    t += dt; // pourquoi si dt plus petit, les fps en mode image diminue fortement ?
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

