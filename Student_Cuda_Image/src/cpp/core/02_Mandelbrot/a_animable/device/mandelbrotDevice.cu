#include "hip/hip_runtime.h"
#include "Thread2D.h"
#include "cudas.h"

#include "real.h"
#include "MandelbrotMath.h"

#include "DomaineMath_GPU.h"
#include "Indices_GPU.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

__global__ void mandelbrot(uchar4* tabPixelsGM , uint w , uint h , DomaineMath domaineMath , int n)
    {
    MandelbrotMath mandelbrotMath(n);
    const int TID = Thread2D::tid();
    const int NB_THREAD = Thread2D::nbThread();
    const int WH = w * h;
    int pixelI, pixelJ;
    double x = 0, y = 0;
    int s = TID;
    while(s<WH){
	Indices::toIJ(s, w, &pixelI, &pixelJ);
	domaineMath.toXY(pixelI, pixelJ, &x, &y);

	mandelbrotMath.colorXY(&tabPixelsGM[s], (real)x, (real)y);
	s+= NB_THREAD;
        }
    // appeler colorXY

    // Probleme : Choix a faire pour le type de (x,y) :
    //
    //			- Travailler en float pour plus de performance?
    //			- Travailler en double pour pouvoir zoomer plus loin dans le fractale?
    //
    // Solution : Travailler avec le typedef real definit dans "real.h"
    //		  Specifier dans "real.h" si vous souhaitez travailler en float ou en double
    //	          Ainsi on passe facilement et proprement d'une version float a une version double
    //
    // Note : domaineMath.toXY travaille lui toujours en double pour vous permettre les deux possibilites ci-dessus
    //
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

