#include "hip/hip_runtime.h"
#include "Mandelbrot.h"

#include <iostream>
#include <assert.h>

#include "Hardware.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void mandelbrot(uchar4* tabPixelsGM, uint w, uint h,DomaineMath domaineMath, int n);

/*--------------------------------------*\
 |*		private		 	*|
 \*-------------------------------------*/

static string titre();

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Mandelbrot::Mandelbrot(const Grid& grid , uint w , uint h , const DomaineMath& domaineMath ,bool isVerbose, int nMin , int nMax) :
	Animable_I<uchar4>(grid, w, h, titre(), domaineMath,isVerbose), // super classe
	variateurT(Interval<int>(nMin, nMax), 1), // tools animation
	nMin(nMin), // Inputs animation
	nMax(nMax) // Inputs animation
    {
    // Tools
    this->t = nMin;
    }

Mandelbrot::~Mandelbrot()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 */
void Mandelbrot::process(uchar4* tabPixelsGM , uint w , uint h , const DomaineMath& domaineMath)
    {
    int n = t;
    mandelbrot<<<Animable_I::dg,Animable_I::db>>>(tabPixelsGM , w , h , domaineMath , n);
    }

/**
 * Override
 * Call periodicly by the API
 */
void Mandelbrot::animationStep()
    {
    this->t = variateurT.varierAndGet();
    }

string titre()
    {
    return "Mandelbrot-Cuda-uchar4-" + realToString();
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

