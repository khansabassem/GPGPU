#include "hip/hip_runtime.h"
#include <iostream>
#include "Thread2D.h"
#include "cudas.h"
#include "Indices_GPU.h"
#include "Interval_GPU.h"
#include "Calibreur_GPU.h"
#include "Colors_GPU.h"
using std::cout;
using std::endl;

#include "heatReal.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void heatTransferToScreenImageHSB(uchar4* ptrDevPixels , real* ptrImageInput , int w , int h);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void heatTransferToScreenImageHSB(uchar4* ptrDevPixels , real* ptrImageInput , int w , int h)
    {
    const int WH = w * h;

    const int NB_THREAD = Thread2D::nbThread();
    const int TID = Thread2D::tid();

    int s = TID;

    real heatMax = 1.0;
    real heatMin = -0.2;
    real hueMax = 0;
    real hueMin = 0.66;

    real pente = (hueMax - hueMin) / (heatMax - heatMin);
    real translation = hueMin - pente * heatMin;
    while (s < WH)
	{
	float hue = ptrImageInput[s] * pente + translation;
	uchar4 p;
	gpu::Colors::HSB_TO_RVB(hue, 1, 1, &p.x, &p.y, &p.z);
	ptrDevPixels[s] = p;
	s += NB_THREAD;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
