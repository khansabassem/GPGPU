#include "hip/hip_runtime.h"
#include <iostream>
#include "Thread2D.h"
#include "cudas.h"
#include "Indices_GPU.h"

using std::cout;
using std::endl;

#include "heatReal.h"


/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void diffusion(real* ptrImageInput , real* ptrImageOutput , int w , int h);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ real D_operator(real* ptrImage , int s , int w , int cardinalCoefficientNS , int cardinalCoefficientWE)
    {
    const real k = 0.21;
    const real adjustedWindNorth = ((real)1 + ((real)cardinalCoefficientNS / (real)100));
    const real adjustedWindEAST = ((real)1 + ((real)cardinalCoefficientWE / (real)100.0));

    const real south = ptrImage[s + w] * adjustedWindNorth;
    const real north = ptrImage[s - w] / adjustedWindNorth;
    const real east = ptrImage[s + 1] / adjustedWindEAST;
    const real west = ptrImage[s - 1] * adjustedWindEAST;
    const real quadri = 4;

    return ptrImage[s] + k * (south + north + east + west - quadri * ptrImage[s]);
    }

__global__ void heatTransferDiffusion(real* ptrImageInput , real* ptrImageOutput , int w , int h , int cardinalCoefficientNS , int cardinalCoefficientWE)
    {
    const int WH = w * h;

    const int NB_THREAD = Thread2D::nbThread();
    const int TID = Thread2D::tid();

    int s = TID;

    while (s < WH)
	{
	// on n'applique pas la diffusion sur les bords
	if (s > w && s < WH - w && (w - 1) % s != 0 && w % s != 0)
	    ptrImageOutput[s] = D_operator(ptrImageInput, s, w, cardinalCoefficientNS, cardinalCoefficientWE);

	s += NB_THREAD;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
