#include "hip/hip_runtime.h"
#include <iostream>
#include "Thread2D.h"
#include "cudas.h"
#include "Indices_GPU.h"

using std::cout;
using std::endl;

#include "heatReal.h"

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void heatTransferEcrasement(real* ptrImageInOutput , int w , int h);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__global__ void heatTransferEcrasement(real* ptrImageInOutput , real* ptrImageHeater , int w , int h)
    {
    const int WH = w * h;

    const int NB_THREAD = Thread2D::nbThread();
    const int TID = Thread2D::tid();

    int s = TID;

    while (s < WH)
	{
	if (ptrImageHeater[s] != (real)0)
	    ptrImageInOutput[s] = ptrImageHeater[s];
	s += NB_THREAD;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
