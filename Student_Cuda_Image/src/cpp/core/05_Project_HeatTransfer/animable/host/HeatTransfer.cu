#include "hip/hip_runtime.h"
#include "HeatTransfer.h"
#include "GM.h"
#include "HM.h"
#include <iostream>
#include <assert.h>
#include "Hardware.h"
#include "HeaterPoint.h"
#include "WindDirection.h"
#include <list>
using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void heatTransferDiffusion(real* ptrImgInputOutput, real* ptrImgHeater, int w, int h,
	int cardinalCoefficientNS, int cardinalCoefficientWE);
extern __global__ void heatTransferEcrasement(real* ptrImgInput, real* ptrImgOutput, int w, int h);
extern __global__ void heatTransferToScreenImageHSB(uchar4* ptrDevPixels, real* ptrImgInput, int w, int h);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

HeatTransfer::HeatTransfer(const Grid& grid , uint w , uint h , float dt , bool isVerbose , HeaterPoint* ptrHeatersQueue , int* ptrWindStrength ,
	WindDirection* ptrWindDirection) :
	Animable_I<uchar4>(grid, w, h, "HeatTransfer-Cuda-uchar4", isVerbose) // super classe
    {
    this->w = w;
    this->h = h;

    // Animation
    this->dt = dt;
    this->t = 0; // protected dans Animable
    this->NB_ITERATION_AVEUGLE = 50;
    this->currentIterationNumber = 0;
    this->heaterPointQueue = ptrHeatersQueue;
    this->windStrength = ptrWindStrength;
    this->windDirection = ptrWindDirection;
    size_t size = this->w * this->h * sizeof(real);

    manageMemory(size);
    initializeImages();
    copyImage(size);
    }

HeatTransfer::~HeatTransfer()
    {
    // rien
    }

HeaterPoint* HeatTransfer::getHeaterQueue()
    {
    return this->heaterPointQueue;
    }

int* HeatTransfer::getWindStrength()
    {
    return this->windStrength;
    }

WindDirection* HeatTransfer::getWindDirection()
    {
    return this->windDirection;
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car image pas zoomable
 */
void HeatTransfer::process(uchar4* tabPixelsGM , uint w , uint h , const DomaineMath& domaineMath)
    {
    real* ptrImageInput = NULL;
    real* ptrImageOutput = NULL;
    if ((this->currentIterationNumber % 2) == 0)
	{
	ptrImageInput = ptrDevImageA;
	ptrImageOutput = ptrDevImageB;
	}
    else
	{
	ptrImageInput = ptrDevImageB;
	ptrImageOutput = ptrDevImageA;
	}

    // Heaters
    if (heaterPointQueue[0].x != 0 || heaterPointQueue[0].y != 0)
	{
	const int pixelToHeat = heaterPointQueue[0].y * this->w + heaterPointQueue[0].x;
	imageHeater[pixelToHeat] = (real)1;

	GM::memcpyHToD(&(ptrDevImageHeater[pixelToHeat]), &(imageHeater[pixelToHeat]), sizeof(real));
	heaterPointQueue[0].x = 0;
	heaterPointQueue[0].y = 0;
	}

    int windStrengthValue = *windStrength;
    /*
     * By default, cardinalCoefficients contains 0 values, where:
     *
     * Index 0 = NORTH
     * Index 1 = EAST
     * Index 2 = SOUTH
     * Index 3 = WEST
     */
    int cardinalCoefficients[2] =
	{
	};
    WindDirection windDirectionValue = *windDirection;
    switch (windDirectionValue)
	{
    case WindDirection::NORTH:
	cardinalCoefficients[0] = windStrengthValue;
	break;
    case WindDirection::EAST:
	cardinalCoefficients[1] = windStrengthValue;
	break;
    case WindDirection::SOUTH:
	cardinalCoefficients[0] = windStrengthValue * -1;
	break;
    case WindDirection::WEST:
	cardinalCoefficients[1] = windStrengthValue * -1;
	break;
    default:
	// nothing to do
	break;
	}

    heatTransferDiffusion<<<dg,db>>>(ptrImageInput, ptrImageOutput, this->w, this->h,
	    cardinalCoefficients[0],
	    cardinalCoefficients[1]);

    heatTransferEcrasement<<<dg,db>>>(ptrImageOutput, ptrDevImageHeater, this->w, this->h);

    //display ??
    if (currentIterationNumber % NB_ITERATION_AVEUGLE == 0)
	{
    heatTransferToScreenImageHSB<<<dg,db>>>(tabPixelsGM, ptrImageOutput, this->w, this->h);
    }

currentIterationNumber++;
}

/**
 * Override
 * Call periodicly by the API
 */
void HeatTransfer::animationStep()
{
t += dt;
}

void HeatTransfer::manageMemory(size_t size)
{
//Global Memory
this->ptrDevImageInit = NULL;
this->ptrDevImageHeater = NULL;
this->ptrDevImageA = NULL;
this->ptrDevImageB = NULL;

GM::malloc(&ptrDevImageInit, size);
GM::malloc(&ptrDevImageHeater, size);
GM::malloc0(&ptrDevImageA, size);
GM::malloc0(&ptrDevImageB, size);
}

void HeatTransfer::copyImage(size_t size)
{
GM::memcpyHToD(ptrDevImageInit, this->imageInit, size);
GM::memcpyHToD(ptrDevImageHeater, this->imageHeater, size);
}

void HeatTransfer::initializeImages()
{
imageHeater = new real[this->w * this->h];
imageInit = new real[this->w * this->h];
}

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

