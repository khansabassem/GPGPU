#include "hip/hip_runtime.h"
#include "SphereCreator.h"

#include <iostream>
#include <assert.h>

#include "Aleas.h"
#include "Maths.h"
#include "HM.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

static bool IS_MEMORY_DMA = true;

/*--------------------------------------*\
 |*		Constructeur		*|
 \*-------------------------------------*/

/**
 * nbSpheres>=4
 */
SphereCreator::SphereCreator(int nbSpheres , int w , int h , int bord) : //
	nbSpheres(nbSpheres), //
	w(w), //
	h(h), //
	bord(bord)

    {
    assert(nbSpheres >= 4);

    if (!IS_MEMORY_DMA)
	{
	this->tabSphere = new Sphere[nbSpheres]; // required default constructeur of type : sphere(void)
	}
    else
	{
	HM::malloc(&tabSphere, nbSpheres * sizeof(Sphere));
	}

    createSphereAll();
    }

SphereCreator::~SphereCreator()
    {
    if (!IS_MEMORY_DMA)
	{
	delete[] tabSphere;
	}
    else
	{
	HM::free(tabSphere);
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

Sphere* SphereCreator::getTabSphere()
    {
    return tabSphere;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

void SphereCreator::createSphereAll(void)
    {
    createSphereBack();
    createSphereFront();
    createSphereAleatoire();
    }

/**
 * La premiere, centrée, très grande, au fond
 */
void SphereCreator::createSphereBack(void)
    {
    float3 centre;

    centre.x = h / 2;
    centre.y = w / 2;
    centre.z = ZMAX * 2; // plus grand que ZMAX

    float rayon = std::min(w / 2, h / 2);
    float hue01 = 1;

    tabSphere[0] = Sphere(centre, rayon, hue01);
    }

/**
 * Les dernieres, centrées, très petite, très proche
 */
void SphereCreator::createSphereFront(void)
    {
    float hue01 = 1;
    float rayon = std::min(w / 10, h / 10); // disons
    float z = ZMIN - (rayon + rayonMax());

    float dhue = 0.2;
    float dz = 40;
    float drayon = dz / 2;

    for (int i = nbSpheres - 3; i < nbSpheres; i++)
	{
	float3 centre;

	z -= dz;
	rayon -= drayon;
	hue01 -= dhue;

	centre.x = h / 2;
	centre.y = w / 2;
	centre.z = z;

	tabSphere[i] = Sphere(centre, rayon, hue01); // Warning : sphere stack copier dans sphere heap, ok car pas ptr dans Sphere
	}
    }

/**
 * Toute sauf la premiere
 */
void SphereCreator::createSphereAleatoire(void)
    {
    Aleas aleas;

    for (int i = 1; i < nbSpheres - 3; i++)
	{
	float3 centre;

	centre.x = aleas.uniformeAB(0 + bord, h - bord);
	centre.y = aleas.uniformeAB(0 + bord, w - bord);
	centre.z = aleas.uniformeAB(ZMIN, ZMAX);

	float rayon = aleas.uniformeAB(20, rayonMax());
	float hue01 = aleas.uniforme01();

	tabSphere[i] = Sphere(centre, rayon, hue01); // Warning : sphere stack copier dans sphere heap, ok car pas ptr dans Sphere
	}
    }

int SphereCreator::rayonMax()
    {
    return w / 10;
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

