#include "hip/hip_runtime.h"
#include "Raytracing.h"

#include <iostream>

#include "MM.h"
#include "GM.h"
#include <assert.h>
#include "SphereCreator.h"
#include "nbSphere.h"
#include "Bandwidth.h"
#include "ChronoFactory.h"

using std::cout;
using std::endl;
using std::string;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

__global__ void kernelRaytacingGM(uchar4* tabPixelsGM , uint w , uint h , float t , Sphere* tabSpheresGM , int nbSpheres);
__global__ void kernelRaytacingSM(uchar4* tabPixelsGM , uint w , uint h , float t , Sphere* tabSpheresGM , int nbSpheres);
__global__ void kernelRaytacingCM(uchar4* tabPixelsGM , uint w , uint h , float t , int nbSpheres);
__global__ void kernelRaytacingCM2SM(uchar4* tabPixelsGM , uint w , uint h , float t , int nbSpheres);

__host__ void uploadToCM(Sphere* tabSpheres , int nbSpheres);

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

static string titre(const MemoryType& memoryType);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur		*|
 \*-------------------------------------*/

Raytracing::Raytracing(const Grid& grid , uint w , uint h , float dt , bool isVerbose , int nbSpheres , const MemoryType& memoryType) :
	Animable_I<uchar4>(grid, w, h, titre(memoryType), isVerbose), // super classe
	memoryType(memoryType), //
	dt(dt), //
	nbSpheres(nbSpheres)
    {
    // Tools
    this->sizeSpheres = nbSpheres * sizeof(Sphere);
    this->t = 0;

    // spheres
	{
	Chrono* ptrChrono = ChronoFactory::create();
	SphereCreator sphereCreator(nbSpheres, w, h);
	ptrChrono->stop();
	if (isVerbose)
	    {
	    cout << "\n[SphereCreator] : " << *ptrChrono << endl;
	    }
	delete ptrChrono;

	uploadToDevice(sphereCreator.getTabSphere());
	} // SphereCreator depiler, donc detruit, tabSphere cote host detruit!
    }

Raytracing::~Raytracing()
    {
    // TODO Raytracing GPU MemoryManagement free
    switch (memoryType)
	{
    case GM:
	{
	GM::free(tabSpheresGM);

	break;
	}
    case CM:
	{
	// Indication: rien a detruire pour la CM
	// Note : la duree de vie pour la CM est une duree processus.

	break;
	}
    case SM:
	{
	// Indication: rien a detruire pour la SM
	// Note : la duree de vie pour la SM est une duree de vie de kernel, seulement!

	break;
	}
    case CM2SM:
	{
	// Indication: rien a detruire pour la SM et CM

	break;
	}
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

/**
 * Override
 */
void Raytracing::process(uchar4* tabPixelsGM , uint w , uint h , const DomaineMath& domaineMath)
    {

    switch (memoryType)
	{
    case GM:
	{
	kernelRaytacingGM<<<dg,db>>>(tabPixelsGM , w , h , t , tabSpheresGM , nbSpheres);
	// Call the kernel kernelRaytacingGM (prototype line 20, about)
	break;
	}
    case CM:
	{
	kernelRaytacingCM<<<dg,db>>>(tabPixelsGM , w , h , t, nbSpheres);
	break;
	}
    case SM:
	{
	size_t arraySphereSize = nbSpheres * sizeof(Sphere);
	kernelRaytacingSM<<<dg,db, arraySphereSize>>>(tabPixelsGM , w , h , t , tabSpheresGM , nbSpheres);
	break;
	}
    case CM2SM:
	{
	size_t arraySphereSize = nbSpheres * sizeof(Sphere);
	kernelRaytacingCM2SM<<<dg,db, arraySphereSize>>>(tabPixelsGM , w, h, t, nbSpheres);
	break;
	}
	}
    }

/**
 * Override
 */
void Raytracing::animationStep()
    {
    t += dt;
    }

/**
 * Override
 */
double Raytracing::getInputGO()
    {
    return NB_SPHERE * sizeof(Sphere) / ((double)1024 * (double)1024 * (double)1024);
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

void Raytracing::uploadToDevice(Sphere* tabSpheres)
    {
    Bandwidth bandwidth(sizeSpheres, "\n" + titre(memoryType) + " : Host -> Device :");

    switch (memoryType)
	{
    case GM:
	{
	// TODO Raytracing GM uploadToDevice
	// But : copier les spheres en GM
	// MM pout la GM ( malloc et memcpy)
	// Utiliser la classe GM
	// step 0
	GM::malloc(&tabSpheresGM, sizeSpheres); // pourquoi un adresse de pointeur
	GM::memcpyHToD(tabSpheresGM, tabSpheres, sizeSpheres);

	break;
	}
    case CM:
	{
	// TODO Raytracing CM uploadToDevice
	// But : copier les spheres en CM
	// Utiliser la methode importer uploadToCM et pister là!
	uploadToCM(tabSpheres , nbSpheres);

	break;
	}
    case SM:
	{
	// TODO Raytracing SM uploadToDevice

	// Indication:
	//		Coter device, on copie GM to SM
	//		Il faut donc d'abord copier les spheres sur le device!
	//		Le code est donc le meme que GM
	GM::malloc(&tabSpheresGM, sizeSpheres); // pourquoi un adresse de pointeur
	GM::memcpyHToD(tabSpheresGM, tabSpheres, sizeSpheres);

	break;
	}
    case CM2SM:
	{
	// Indication:
	//		Coter device, on copie CM to SM
	//		Il faut donc d'abord copier les spheres en CM
	//		Le code est donc le meme que CM
	uploadToCM(tabSpheres , nbSpheres);
	break;
	}
	}

    if (isVerbose)
	{
	cout << endl << bandwidth << endl;
	}
    }

string titre(const MemoryType& memoryType)
    {
    switch (memoryType)
	{
    case GM:
	return "Raytracing-GM-uchar4";

    case CM:
	return "Raytracing-CM-uchar4";

    case SM:
	return "Raytracing-SM-uchar4";

    case CM2SM:
	return "Raytracing-CM2SM-uchar4";

    default:
	{
	assert(false);
	return "ERROR";
	}
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

