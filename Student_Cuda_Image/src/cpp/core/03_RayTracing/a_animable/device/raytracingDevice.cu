#include "hip/hip_runtime.h"
#include <assert.h>

#include "Thread2D.h"
#include "cudas.h"
#include "Indices_GPU.h"

#include "Sphere.h"
#include "nbSphere.h"
#include "SM.h"
#include "raytracingCM.h"
#include "RaytracingMath.h"
#include "DomaineMath_GPU.h"

using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

static __device__ void work(uchar4* tabPixelsGM, uint w, uint h, float t, Sphere* tabSpheresDev, int nbSpheres);
static __device__ void copyDevToSM(Sphere* ptrSpheresGM , Sphere* ptrSpheresSM , int size);
/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

__global__ void kernelRaytacingGM(uchar4* tabPixelsGM , uint w , uint h , float t , Sphere* tabSpheresGM , int nbSpheres)
    {
    // TODO Raytracing GPU GM

    // Indications :
    //		(I1) Call methode work with good input
    // 		(I2) work contain the algo
    //		(I3) The algo is the same with the GM,CM,SM of TP Ractracing
    work(tabPixelsGM, w, h, t, tabSpheresGM, nbSpheres);

    }

__global__ void kernelRaytacingSM(uchar4* tabPixelsGM , uint w , uint h , float t , Sphere* tabSpheresGM , int nbSpheres)
    {
    // TODO Raytracing GPU SM

    // Indications :
    //		(I1) Copier les sphere de GM to SM
    // 		(I2) Call work with good input
    //		(I3) Implementer une methode copyGMtoSM
    extern __shared__ Sphere tabSpheresSM[];
    copyDevToSM(tabSpheresGM, tabSpheresSM, nbSpheres);
    __syncthreads();
    work(tabPixelsGM, w, h, t, tabSpheresSM, nbSpheres);
    }

__device__ void copyDevToSM(Sphere* ptrSpheresGM , Sphere* ptrSpheresSM , int size)
    {
    const int TID_LOCAL = Thread2D::tidLocal();
    const int NB_THREAD_BLOCK = Thread2D::nbThreadBlock();
    int s = TID_LOCAL;
    while (s < size)
	{
	ptrSpheresSM[s] = ptrSpheresGM[s];
	s += NB_THREAD_BLOCK;
	}
    }

__global__ void kernelRaytacingCM(uchar4* tabPixelsGM , uint w , uint h , float t , int nbSpheres)
    {
    work(tabPixelsGM, w, h, t, TAB_SPHERES_CM, nbSpheres);
    // Indications :
    //		(I1) call work with good input
    // 		(I2) TAB_SPHERES_CM est une variable globale a ce fichier! (Voir le debut de ce fichier, include rayTracingCM.h)
    }

__global__ void kernelRaytacingCM2SM(uchar4* tabPixelsGM , uint w , uint h , float t , int nbSpheres)
    {
    // TODO Raytracing GPU SM_BY_CM
    extern __shared__ Sphere tabSpheresSM[];
    copyDevToSM(TAB_SPHERES_CM , tabSpheresSM , nbSpheres);
    __syncthreads();
    work(tabPixelsGM, w, h, t, tabSpheresSM, nbSpheres);
    // Indications :
    //		(I1) Copier les sphere de CM to SM
    // 		(I2) Call work with good input
    //		(I3) Implementer une methode copyCMtoSM
    //		     ou : Tip : renommer copyGMtoSM en copyDevToSM et utiliser copyDevToSM avec les bons inputs
    }

/*--------------------------------------*\
 |*		private			*|
 \*-------------------------------------*/

/**
 * Methode commune au 3 kernel ci-dessus.
 * Ici on ne sait pas si derriere tabSpheresDev, c'est
 * 	- de la GM?
 * 	- de la SM?
 * 	- de la CM?
 * Pas d'importance, c'est un pointeur et on travail avec!
 */
__device__ void work(uchar4* tabPixelsGM , uint w , uint h , float t , Sphere* tabSpheresDev , int nbSpheres)
    {
    RaytracingMath raytracingMath(tabSpheresDev, nbSpheres, t);
    const int TID = Thread2D::tid();
    const int NB_THREAD = Thread2D::nbThread();
    const int WH = w * h;
    int pixelI, pixelJ;
    int s = TID;
    while (s < WH)
	{
	Indices::toIJ(s, w, &pixelI, &pixelJ);
	raytracingMath.colorXY(&tabPixelsGM[s], pixelI, pixelJ);
	s += NB_THREAD;
	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

