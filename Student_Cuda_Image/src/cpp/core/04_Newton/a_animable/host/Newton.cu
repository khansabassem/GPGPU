#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "Newton.h"
#include <iostream>
#include <assert.h>
#include "Maths.h"
#include "Hardware.h"
/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void newton(uchar4* tabPixelsGM, uint w, uint h, DomaineMath domaineMath, uint n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Newton::Newton(const Grid& grid , uint w , uint h , uint nMin, uint nMax, const DomaineMath& domaineMath) :
	Animable_I<uchar4>(grid, w, h, "Newton_Cuda_RGBA_uchar4", domaineMath),
	variateurN(Interval<int>(nMin, nMax), 1) //
//	variateurN(Interval<int>(10,1000),1)
//	n(n)

    {

    // Inputs
    this->w = w;
    this->h = h;
//    this->ptrDomaineMathInit = domaineMath;

    // get dg, db
    this->dg = grid.dg;
    this->db= grid.db;

    // Tools
    //Outputs
    this->title = "Newton_CUDA (Zoomable)";
    }

Newton::~Newton()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 */
void Newton::process(uchar4* tabPixelsGM , uint w , uint h , const DomaineMath& domaineMath)
    {
    //time elapsed for kernel calls, perf indicator
    newton<<<dg,db>>>(tabPixelsGM,w,h,domaineMath,n);
    // newton<<<w,h>>>(tabPixelsGM,w,h,domaineMath,n);
//    newton<<<Hardware::getMPCount(),Hardware::getCoreCountMP()*16>>>(tabPixelsGM,w,h,domaineMath,n);
//    hipDeviceSynchronize();
//    clock_t end = clock();
//    printf("n=%d    Time Elapsed: %.3lf ms\n", n, (double)(end-begin)/CLOCKS_PER_SEC*1000.0);
    //printf("n=%d    Time Elapsed: %.3lf ms\n", n, (double)(end-begin)/CLOCKS_PER_SEC*1000.0);
    }

/**
 * Override
 * Call periodicly by the API
 */
void Newton::animationStep()
{
this->n = variateurN.varierAndGet()/10; // in [0,2pi]
}


/*--------------*\
 |*	get	 *|
 \*--------------*/

/**
 * Override
 */
//DomaineMath* Newton::getDomaineMathInit(void)
//{
//return ptrDomaineMathInit;
//}

/**
 * Override
 */
float Newton::getAnimationPara(void)
{
return n;
}

/**
 * Override
 */
int Newton::getW(void)
{
return w;
}

/**
 * Override
 */
int Newton::getH(void)
{
return h;
}

/**
 * Override
 */
string Newton::getTitle(void)
{
return title;
}

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

