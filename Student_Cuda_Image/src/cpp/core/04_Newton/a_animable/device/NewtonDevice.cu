#include "hip/hip_runtime.h"
#include "Thread2D.h"
#include "Thread1D.h"
#include "cudas.h"

#include "NewtonMath.h"

#include "Indices_GPU.h"
#include "DomaineMath_GPU.h"

//by me
#include "Hardware.h"
using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void newton(uchar4* tabPixelsGM, uint w, uint h, DomaineMath domaineMath, uint n);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

//by me
__global__ void newton(uchar4* ptrDevPixels, uint w, uint h, DomaineMath domaineMath, uint n)
    {
    float epsilon =0.1f*powf(0.90f,n);// 0.1f;
    NewtonMath newtonMath = NewtonMath(epsilon, n);

    const int TID = Thread2D::tid();
    const int NB_THREAD = Thread2D::nbThread();

    const int WH = w * h;

    uchar4 color;

    double x;
    double y;

    int pixelI;
    int pixelJ;

    int s = TID;
    while (s < WH)
	{
	Indices::toIJ(s, w, &pixelI, &pixelJ); // update (pixelI, pixelJ)

	// (i,j) domaine ecran
	// (x,y) domaine math
	domaineMath.toXY(pixelI, pixelJ, &x, &y); //  (i,j) -> (x,y)

	newtonMath.colorXY(&color, x, y); // update color

	ptrDevPixels[s] = color;

	s += NB_THREAD;
	}
//__global__ void newton(uchar4* tabPixelsGM, uint w, uint h, DomaineMath domaineMath, uint n)
//    {
//    //by me
//    float epsilon = 0.1f*powf(0.90f,n);//n varies with time so does epsilon
//    NewtonMath newtonMath = NewtonMath(epsilon, n/2);
//// until here by me
//    const int TID = Thread2D::tid();//();
//    const int NB_THREAD = Thread2D::nbThread();
//    const int WH = w * h;
//    const int LEAP = blockDim.x * gridDim.x;
//// by    me4
//    uchar4 color;
//// until here
//    double x;
//    double y;
//
//    // added 2020.8.26
//    int idx = blockIdx.x * blockDim.x + threadIdx.x;
//    int i; // in [0,h[
//    int j; // in [0,w[
//
//    int s = TID;
//    while (s < WH)
////    while (idx < WH)
//	{
//	Indices::toIJ(s, w, &i, &j); // update (i, j)// ok
////
//	i=blockIdx.x;
//	j=threadIdx.x;
//
//	i = idx / w;
//	j = idx % w;
//	// (i,j) domaine ecran
//	// (x,y) domaine math
//	domaineMath.toXY(i, j, &x, &y); //  (i,j) -> (x,y) ok
//	newtonMath.colorXY(&color, x, y); // update color
//	tabPixelsGM[idx] = color;

//	 s += NB_THREAD;
////	idx += LEAP;
//	}


    }

/*-----------------------------------a-----------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

